#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "CImg.h"
using namespace cimg_library;

#define MIN(a,b) ((a < b) ? a : b)
#define MAX(a,b) ((a > b) ? a : b)

hipError_t redFilterCUDA(const char *fname);
hipError_t greenFilterCUDA(const char *fname);
hipError_t blueFilterCUDA(const char *fname);
hipError_t yellowFilterCUDA(const char *fname);
hipError_t magentaFilterCUDA(const char *fname);
hipError_t cyanFilterCUDA(const char *fname);
hipError_t grayscaleLightnessCUDA(const char *fname);
hipError_t grayscaleAverageCUDA(const char *fname);
hipError_t grayscaleLuminosityRCUDA(const char *fname);
hipError_t grayscaleLuminosityGCUDA(const char *fname);
hipError_t grayscaleLuminosityBCUDA(const char *fname);

/* Color filters should be applied to grayscaled images.
	It is possible to create a grayscale image and then apply the filter. */

__global__ void redFilterKernel(unsigned char *img, int w, int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	img[(y + h) * w + x] = 0;
	img[(y + h + h) * w + x] = 0;
}

__global__ void greenFilterKernel(unsigned char *img, int w, int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	img[y * w + x] = 0;
	img[(y + h + h) * w + x] = 0;
}

__global__ void blueFilterKernel(unsigned char *img, int w, int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	img[y * w + x] = 0;
	img[(y + h) * w + x] = 0;
}

__global__ void yellowFilterKernel(unsigned char *img, int w, int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	img[(y + h + h) * w + x] = 0;
}

__global__ void magentaFilterKernel(unsigned char *img, int w, int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	img[(y + h) * w + x] = 0;
}

__global__ void cyanFilterKernel(unsigned char *img, int w, int h)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	img[y * w + x] = 0;
}

__global__ void grayscaleLightnessKernel(unsigned char *img, int w, int h) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	int R = img[y * w + x];
	int G = img[(y + h) * w + x];
	int B = img[(y + h + h) * w + x];

	int gray = (MAX(R, MAX(G, B)) + MIN(R, MIN(G, B))) / 2;

	img[y * w + x] = img[(y + h) * w + x] = img[(y + h + h) * w + x] = gray;
}

__global__ void grayscaleAverageKernel(unsigned char *img, int w, int h) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	// R, G, B values for pixel are assigned average of said R, G, B values;
	img[y * w + x] = img[(y + h) * w + x] = img[(y + h + h) * w + x]
		= (img[y * w + x] + img[(y + h) * w + x] + img[(y + h + h) * w + x]) / 3;
}

/* General Luminosity:
	(0.21 R + 0.72 G + 0.07 B) / 3
	(people are more sensitive to green) */

__global__ void grayscaleLuminosityRKernel(unsigned char *img, int w, int h) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	// R, G, B values for pixel are assigned scaled average of said R, G, B values;
	img[y * w + x] = img[(y + h) * w + x] = img[(y + h + h) * w + x]
		= (img[y * w + x] * 0.72 + img[(y + h) * w + x] * 0.21 + img[(y + h + h) * w + x] * 0.07) / 3;
}

__global__ void grayscaleLuminosityGKernel(unsigned char *img, int w, int h) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	// R, G, B values for pixel are assigned scaled average of said R, G, B values;
	img[y * w + x] = img[(y + h) * w + x] = img[(y + h + h) * w + x]
		= (img[y * w + x] * 0.21 + img[(y + h) * w + x] * 0.72 + img[(y + h + h) * w + x] * 0.07) / 3;
}

__global__ void grayscaleLuminosityBKernel(unsigned char *img, int w, int h) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= w || y >= h) return;

	// R, G, B values for pixel are assigned scaled average of said R, G, B values;
	img[y * w + x] = img[(y + h) * w + x] = img[(y + h + h) * w + x]
		= (img[y * w + x] * 0.07 + img[(y + h) * w + x] * 0.21 + img[(y + h + h) * w + x] * 0.72) / 3;
}

int checkFunctionCudaSuccess(hipError_t status, char *func)
{
	int rc = 0;
	if (status != hipSuccess)
	{
		fprintf(stderr, "%s failed!", func);
		rc = 1;
	}
	return rc;
}

int main(int argc, char *argv[])
{
	hipError_t cudaStatus;

    // Adds red filter in parallel
    cudaStatus = redFilterCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "redFilterCUDA"))
	{
		return 1;
	}

	// Adds green filter in parallel
	cudaStatus = greenFilterCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "greenFilterCUDA"))
	{
		return 1;
	}

	// Adds blue filter in parallel
	cudaStatus = blueFilterCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "blueFilterCUDA"))
	{
		return 1;
	}

	// Adds yellow filter in parallel
	cudaStatus = yellowFilterCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "yellowFilterCUDA"))
	{
		return 1;
	}

	// Adds magenta filter in parallel
	cudaStatus = magentaFilterCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "magentaFilterCUDA"))
	{
		return 1;
	}

	// Adds cyan filter in parallel
	cudaStatus = cyanFilterCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "cyanFilterCUDA"))
	{
		return 1;
	}

	// Turn image grayscale using Lightness method in parallel
	cudaStatus = grayscaleLightnessCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "grayscaleLightnessCUDA"))
	{
		return 1;
	}

	// Turn image grayscale using Average method in parallel
	cudaStatus = grayscaleAverageCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "grayscaleAverageCUDA"))
	{
		return 1;
	}

	// Turn image grayscale using Luminosity R method in parallel
	cudaStatus = grayscaleLuminosityRCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "grayscaleLuminosityRCUDA"))
	{
		return 1;
	}

	// Turn image grayscale using Luminosity G method in parallel
	cudaStatus = grayscaleLuminosityGCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "grayscaleLuminosityGCUDA"))
	{
		return 1;
	}

	// Turn image grayscale using Luminosity B method in parallel
	cudaStatus = grayscaleLuminosityBCUDA(argv[1]);
	if (checkFunctionCudaSuccess(cudaStatus, "grayscaleLuminosityBCUDA"))
	{
		return 1;
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	return 0;
}

hipError_t redFilterCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	redFilterKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "redFilterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching redFilterKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/redFilter.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

hipError_t greenFilterCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	greenFilterKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "greenFilterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching redFilterKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/greenFilter.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

hipError_t blueFilterCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	blueFilterKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "blueFilterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching blueFilterKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/blueFilter.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

hipError_t yellowFilterCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	yellowFilterKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "yellowFilterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching yellowFilterKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/yellowFilter.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

hipError_t magentaFilterCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	magentaFilterKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "magentaFilterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching magentaFilterKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/magentaFilter.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

hipError_t cyanFilterCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	cyanFilterKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cyanFilterKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching cyanFilterKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/cyanFilter.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

// Helper function for using CUDA to turn an image grayscale using Lightness method in parallel.
hipError_t grayscaleLightnessCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	grayscaleLightnessKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "grayscaleLightnessKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching removeRKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/grayscaleLightness.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

// Helper function for using CUDA to turn an image grayscale using Average method in parallel.
hipError_t grayscaleAverageCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	grayscaleAverageKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "grayscaleAverageKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching removeRKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/grayscaleAverage.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

// Helper function for using CUDA to turn an image grayscale using Luminosity R method in parallel.
hipError_t grayscaleLuminosityRCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	grayscaleLuminosityRKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "grayscaleLuminosityRKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching removeRKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/grayscaleLuminosityR.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

// Helper function for using CUDA to turn an image grayscale using Luminosity G method in parallel.
hipError_t grayscaleLuminosityGCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	grayscaleLuminosityGKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "grayscaleLuminosityGKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching removeRKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/grayscaleLuminosityG.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}

// Helper function for using CUDA to turn an image grayscale using Luminosity B method in parallel.
hipError_t grayscaleLuminosityBCUDA(const char *fname)
{
	CImg<unsigned char> src(fname);
	int w = src.width();
	int h = src.height();
	unsigned long s = src.size();
	unsigned char *src_ptr = src.data();
	unsigned char *src_kptr;
	
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	// Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&src_kptr, s);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(src_kptr, src_ptr, s, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// Launch kernel for 256 blocks and a multitude of threads based on image size.
	dim3 blkDim(16, 16, 1);
	dim3 grdDim((w + 15) / 16, (h + 15) / 16, 1);
	grayscaleLuminosityBKernel<<<grdDim, blkDim>>>(src_kptr, w, h);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "grayscaleLuminosityBKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching removeRKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(src_ptr, src_kptr, s, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	src.save("./images/grayscaleLuminosityB.bmp");

Error:
    hipFree(src_kptr);

	return cudaStatus;
}
